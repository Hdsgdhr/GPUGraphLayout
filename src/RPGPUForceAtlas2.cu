#include "hip/hip_runtime.h"
/*
 ==============================================================================

 RPGPUForceAtlas2.cu
 Copyright (C) 2016, 2017  G. Brinkmann

 This file is part of graph_viewer.

 graph_viewer is free software: you can redistribute it and/or modify
 it under the terms of version 3 of the GNU Affero General Public License as
 published by the Free Software Foundation.

 graph_viewer is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU Affero General Public License for more details.

 You should have received a copy of the GNU Affero General Public License
 along with graph_viewer.  If not, see <https://www.gnu.org/licenses/>.

 ==============================================================================
*/

#include <stdio.h>
#include <fstream>
#include <chrono>
#include <algorithm>
#include "time.h"

#include "RPGPUForceAtlas2.hpp"
#include "RPBHFA2LaunchParameters.cuh"
#include "RPBHKernels.cuh"
#include "RPFA2Kernels.cuh"

namespace RPGraph
{
    CUDAForceAtlas2::CUDAForceAtlas2(GraphLayout &layout)
    : ForceAtlas2(layout)
    {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (deviceCount == 0)
        {
            fprintf(stderr, "error: No CUDA devices found.\n");
            exit(EXIT_FAILURE);
        }

        // Host initialization and setup //
        nbodies = layout.graph.num_nodes();
        nedges  = layout.graph.num_edges();

        body_pos = (float2 *)malloc(sizeof(float2) * layout.graph.num_nodes());
        body_mass = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        sources  = (int *)  malloc(sizeof(int)   * layout.graph.num_edges());
        targets  = (int *)  malloc(sizeof(int)   * layout.graph.num_edges());
        fx       = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fy       = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fx_prev  = (float *)malloc(sizeof(float) * layout.graph.num_nodes());
        fy_prev  = (float *)malloc(sizeof(float) * layout.graph.num_nodes());

        for (nid_t n = 0; n < layout.graph.num_nodes(); ++n)
        {
            body_pos[n] = {layout.getX(n), layout.getY(n)};
            body_mass[n] = ForceAtlas2::mass(n);
            fx[n] = 0.0;
            fy[n] = 0.0;
            fx_prev[n] = 0.0;
            fy_prev[n] = 0.0;
        }

        int cur_sources_idx = 0;
        int cur_targets_idx = 0;

        // Initialize the sources and targets arrays with edge-data.
        for (nid_t source_id = 0; source_id < layout.graph.num_nodes(); ++source_id)
        {
            for (nid_t target_id : layout.graph.neighbors_with_geq_id(source_id))
            {
                sources[cur_sources_idx++] = source_id;
                targets[cur_targets_idx++] = target_id;
            }
        }

        // GPU initialization and setup //
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        if (deviceProp.warpSize != WARPSIZE)
        {
            printf("Warpsize of device is %d, but we anticipated %d\n", deviceProp.warpSize, WARPSIZE);
            exit(EXIT_FAILURE);

        }
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel1), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
#if __CUDA_ARCH__ < 300
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferL1);
#endif
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(DisplacementKernel), hipFuncCachePreferL1);

        hipGetLastError();  // reset error value

        // Allocate space on device.
        mp_count = deviceProp.multiProcessorCount;
        max_threads_per_block = deviceProp.maxThreadsPerBlock;

        nnodes = std::max(2 * nbodies, mp_count * max_threads_per_block);

        // Round up to next multiple of WARPSIZE
        while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
        nnodes--;

        // child stores structure of the quadtree. values point to IDs.
        cudaCatchError(hipMalloc((void **)&childl,  sizeof(int)   * (nnodes+1) * 4));

        // the following properties, for each node in the quadtree (both internal and leaf)
        cudaCatchError(hipMalloc((void **)&body_massl,   sizeof(float) * nbodies));
        cudaCatchError(hipMalloc((void **)&node_massl,   sizeof(float) * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&body_posl,sizeof(float2) * nbodies));
        cudaCatchError(hipMalloc((void **)&node_posl,    sizeof(float2) * (nnodes+1)));
        // count contains the number of nested nodes for each node in quadtree
        cudaCatchError(hipMalloc((void **)&countl,  sizeof(int)   * (nnodes+1)));
        // start contains ...
        cudaCatchError(hipMalloc((void **)&startl,  sizeof(int)   * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&sortl,   sizeof(int)   * (nnodes+1)));


        cudaCatchError(hipMalloc((void **)&sourcesl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&targetsl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&fxl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fyl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fx_prevl,sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fy_prevl,sizeof(float) * (nbodies)));

        // Used for reduction in BoundingBoxKernel
        cudaCatchError(hipMalloc((void **)&maxxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&maxyl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minyl,   sizeof(float) * mp_count * FACTOR1));

        // Used for reduction in SpeedKernel
        cudaCatchError(hipMalloc((void **)&swgl,    sizeof(float) * mp_count * FACTOR6));
        cudaCatchError(hipMalloc((void **)&etral,   sizeof(float) * mp_count * FACTOR6));

        // Copy host data to device.
        cudaCatchError(hipMemcpy(body_massl, body_mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(body_posl,  body_pos,  sizeof(float2) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));

        // cpy fx, fy , fx_prevl, fy_prevl so they are all initialized to 0 in device memory.
        cudaCatchError(hipMemcpy(fxl, fx,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fyl, fy,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fx_prevl, fx_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fy_prevl, fy_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    }

    void CUDAForceAtlas2::freeGPUMemory()
    {
        hipFree(childl);

        hipFree(body_massl);
        hipFree(node_massl);
        hipFree(body_posl);
        hipFree(node_posl);
        hipFree(sourcesl);
        hipFree(targetsl);
        hipFree(countl);
        hipFree(startl);
        hipFree(sortl);

        hipFree(fxl);
        hipFree(fx_prevl);
        hipFree(fyl);
        hipFree(fy_prevl);

        hipFree(maxxl);
        hipFree(maxyl);
        hipFree(minxl);
        hipFree(minyl);

        hipFree(swgl);
        hipFree(etral);
    }

    CUDAForceAtlas2::~CUDAForceAtlas2()
    {
        free(body_mass);
        free(body_pos);
        free(sources);
        free(targets);
        free(fx);
        free(fy);
        free(fx_prev);
        free(fy_prev);

        freeGPUMemory();
    }

    void CUDAForceAtlas2::benchmark()
    {
        printf("Using %d MPs\n", mp_count);
        const int num_reps = 5;

        float times[11] = {0.0, };
        float time; // to temporarily hold time of a kernel

        const char *kernel_names[11] = {"Gravity", "Attractive", "BoundingBox", "Clear1", "TreeBuilding", "Clear2", "Summarization", "Sort", "Force", "Speed", "Displacement"};

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        auto starttime = std::chrono::high_resolution_clock::now();

        for (int i = 0; i < num_reps; ++i)
        {
            hipEventRecord(start, 0);
            GravityKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g, strong_gravity, body_massl, body_posl, fxl, fyl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[0] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            AttractiveForceKernel<<<mp_count * FACTOR6, THREADS6>>>(nedges, body_posl, fxl, fyl, sourcesl, targetsl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[1] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            BoundingBoxKernel<<<mp_count * FACTOR1, THREADS1>>>(nnodes, nbodies, startl, childl, node_massl, body_posl, node_posl, maxxl, maxyl, minxl, minyl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[2] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            ClearKernel1<<<1024, 1>>>(nnodes, nbodies, childl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[3] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            TreeBuildingKernel<<<mp_count * FACTOR2, THREADS2>>>(nnodes, nbodies, childl, body_posl, node_posl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[4] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            ClearKernel2<<<1024, 1>>>(nnodes, startl, node_massl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[5] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            SummarizationKernel<<<mp_count * FACTOR3, THREADS3>>>(nnodes, nbodies, countl, childl, body_massl, node_massl, body_posl, node_posl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[6] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            SortKernel<<<mp_count * FACTOR4, THREADS4>>>(nnodes, nbodies, sortl, countl, startl, childl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[7] += time;
            cudaCatchError(hipGetLastError());

            float epssq  = 0.05 * 0.05;            // Some sort of softening (eps, squared)
            float itolsq = 1.0f / (theta * theta); // Inverse tolerance, squared
            hipEventRecord(start, 0);
            ForceCalculationKernel<<<mp_count * FACTOR5, THREADS5>>>(nnodes, nbodies, itolsq, epssq, sortl, childl, body_massl, node_massl, body_posl, node_posl, fxl, fyl, k_r);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[8] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            SpeedKernel<<<mp_count * FACTOR1, THREADS1>>>(nbodies, fxl, fyl, fx_prevl, fy_prevl, body_massl, swgl, etral);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[9] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            DisplacementKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, body_posl, fxl, fyl, fx_prevl, fy_prevl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[10] += time;
            cudaCatchError(hipGetLastError());
        }

        hipDeviceSynchronize(); // Not really neccesary given the preceding hipEventSynchronize(stop).
        auto endtime = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> runningtime = (endtime - starttime);
        std::chrono::microseconds runningtime_us = std::chrono::duration_cast<std::chrono::microseconds>(runningtime);

        printf("Benchmarking Results (averaging %d times):\n", num_reps);
        printf("\tkernel durations (us), resolution: ... us.:\n");
        for (int i = 0; i < 11; ++i)
        {
            printf("%s %.4f\n", kernel_names[i], 1000.0 * times[i] / (float)num_reps);
        }
        printf("\n");
        printf("Total %.2f\n", runningtime_us.count() / (float)num_reps);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void CUDAForceAtlas2::doStep()
    {
        int err = 0;

        GravityKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g, strong_gravity, body_massl, body_posl, fxl, fyl);

        AttractiveForceKernel<<<mp_count * FACTOR6, THREADS6>>>(nedges, body_posl, fxl, fyl, sourcesl, targetsl);

        BoundingBoxKernel<<<mp_count * FACTOR1, THREADS1>>>(nnodes, nbodies, startl, childl, node_massl, body_posl, node_posl, maxxl, maxyl, minxl, minyl);

        // Build Barnes-Hut Tree
        // 1.) Set all child pointers of internal nodes (in childl) to null (-1)
        ClearKernel1<<<mp_count, 1024>>>(nnodes, nbodies, childl);
        // 2.) Build the tree
        TreeBuildingKernel<<<mp_count * FACTOR2, THREADS2>>>(nnodes, nbodies, childl, body_posl, node_posl);

        hipDeviceSynchronize();
        cudaCatchError(hipMemcpyFromSymbol(&err, HIP_SYMBOL(errd), sizeof(int), 0, hipMemcpyDeviceToHost));
        if (err != 0)
        {
            fprintf(stderr, "error: An error occurred in TreeBuildingKernel, errd == %d\n", err);
            // exit(EXIT_FAILURE);
        }

        // 3.) Set all cell mass values to -1.0, set all startd to null (-1)
        ClearKernel2<<<mp_count, 1024>>>(nnodes, startl, node_massl);

        // Recursively compute mass for each BH. cell.
        SummarizationKernel<<<mp_count * FACTOR3, THREADS3>>>(nnodes, nbodies, countl, childl, body_massl, node_massl, body_posl, node_posl);

        SortKernel<<<mp_count * FACTOR4, THREADS4>>>(nnodes, nbodies, sortl, countl, startl, childl);

        // Compute repulsive forces between nodes using BH. tree.
        float epssq  = 0.05 * 0.05;            // Some sort of softening (eps, squared)
        float itolsq = 1.0f / (theta * theta); // Inverse tolerance, squared
        ForceCalculationKernel<<<mp_count * FACTOR5, THREADS5>>>(nnodes, nbodies, itolsq, epssq, sortl, childl, body_massl, node_massl, body_posl, node_posl, fxl, fyl, k_r);

        hipDeviceSynchronize();
        cudaCatchError(hipMemcpyFromSymbol(&err, HIP_SYMBOL(errd), sizeof(int), 0, hipMemcpyDeviceToHost));
        if (err != 0)
        {
            fprintf(stderr, "error: An error occurred in ForceCalculationKernel, errd == %d\n", err);
            exit(EXIT_FAILURE);
        }

        SpeedKernel<<<mp_count * FACTOR1, THREADS1>>>(nbodies, fxl, fyl, fx_prevl, fy_prevl, body_massl, swgl, etral);

        DisplacementKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, body_posl, fxl, fyl, fx_prevl, fy_prevl);

        iteration++;
    }

    void CUDAForceAtlas2::retrieveLayoutFromGPU()
    {
        cudaCatchError(hipMemcpy(body_pos, body_posl, sizeof(float2) * nbodies, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
    }

    void CUDAForceAtlas2::sendLayoutToGPU()
    {
        cudaCatchError(hipMemcpy(body_posl, body_pos, sizeof(float2) * nbodies, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
    }

    void CUDAForceAtlas2::sendGraphToGPU()
    {
        cudaCatchError(hipMemcpy(body_massl, body_mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
    }
    
    void CUDAForceAtlas2::sync_layout() 
    {
        retrieveLayoutFromGPU();
        for(nid_t n = 0; n < layout.graph.num_nodes(); ++n)
        {
            layout.setX(n, body_pos[n].x);
            layout.setY(n, body_pos[n].y);
        }
    }
}
