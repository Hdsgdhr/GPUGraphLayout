#include "hip/hip_runtime.h"
/*
 ==============================================================================

 RPCUDAForceAtlas2.cu
 Copyright (C) 2016, 2017  G. Brinkmann

This file is part of graph_viewer.

 graph_viewer is free software: you can redistribute it and/or modify
 it under the terms of version 3 of the GNU Affero General Public License as
 published by the Free Software Foundation.

 graph_viewer is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU Affero General Public License for more details.

 You should have received a copy of the GNU Affero General Public License
 along with graph_viewer.  If not, see <https://www.gnu.org/licenses/>.

 ==============================================================================
*/

#include <stdio.h>
#include <fstream>
#include <chrono>
#include <algorithm>
#include "time.h"

#include "RPCUDAForceAtlas2.hpp"
#include "RPCommon.hpp"

#include "RPCUDALaunchParameters.cuh"
#include "RPBHKernels.cuh"
#include "RPFA2Kernels.cuh"

#include "../lib/pngwriter/src/pngwriter.h"

namespace RPGraph
{
    CUDAFA2Layout::CUDAFA2Layout(UGraph &graph, float width, float height)
    : graph(graph), width(width), height(height)
    {
        // General FA2 code //
        iteration = 0;

        k_g = 1.0;
        k_r = 1.0;

        global_speed = 1.0;
        speed_efficiency = 1.0;
        jitter_tolerance = 1.0;

        k_s = 0.1;
        k_s_max = 10.0;
        theta = 1.0;

        delta = 0.0;

        prevent_overlap = false;
        strong_gravity = false;
        use_barneshut = true;
        use_linlog = false;

        if (!use_barneshut)
        {
            printf("RPCUDAForeceAtlas2 without Barnes-Hut approximation is not implemented yet.\n");
            exit(EXIT_FAILURE);
        }


        // Host initialization and setup //
        nbodies = graph.num_nodes();
        nedges  = graph.num_edges();

        posx     = (float *)malloc(sizeof(float) * graph.num_nodes());
        posy     = (float *)malloc(sizeof(float) * graph.num_nodes());
        mass     = (float *)malloc(sizeof(float) * graph.num_nodes());
        sources  = (int *)  malloc(sizeof(int)   * graph.num_edges());
        targets  = (int *)  malloc(sizeof(int)   * graph.num_edges());
        fx       = (float *)malloc(sizeof(float) * graph.num_nodes());
        fy       = (float *)malloc(sizeof(float) * graph.num_nodes());
        fx_prev  = (float *)malloc(sizeof(float) * graph.num_nodes());
        fy_prev  = (float *)malloc(sizeof(float) * graph.num_nodes());

        for (nid_t n = 0; n <  graph.num_nodes(); ++n)
        {
            posx[n] = get_random(-width/2.0,  width/2.0);
            posy[n] = get_random(-height/2.0, height/2.0);
            mass[n] = graph.degree(n) + 1;
            fx[n] = 0.0;
            fy[n] = 0.0;
            fx_prev[n] = 0.0;
            fy_prev[n] = 0.0;
        }

        int cur_sources_idx = 0;
        int cur_targets_idx = 0;

        // Initialize the sources and targets arrays with edge-data.
        for (nid_t source_id = 0; source_id < graph.num_nodes(); ++source_id)
        {
            for (nid_t target_id : graph.neighbors_with_geq_id(source_id))
            {
                sources[cur_sources_idx++] = source_id;
                targets[cur_targets_idx++] = target_id;
            }
        }

        // GPU initialization and setup //
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        if (deviceProp.warpSize != WARPSIZE)
        {
            printf("Warpsize of device is %d, but we anticipated %d\n", deviceProp.warpSize, WARPSIZE);
            exit(EXIT_FAILURE);

        }
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel1), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferShared);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);
#if __CUDA_ARCH__ < 300
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferL1);
#endif
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(DisplacementKernel), hipFuncCachePreferL1);

        hipGetLastError();  // reset error value

        // Allocate space on device.
        mp_count = deviceProp.multiProcessorCount;
        max_threads_per_block = deviceProp.maxThreadsPerBlock;

        nnodes = std::max(2 * nbodies, mp_count * max_threads_per_block);

        // Round up to next multiple of WARPSIZE
        while ((nnodes & (WARPSIZE-1)) != 0) nnodes++;
        nnodes--;

        // child stores structure of the quadtree. values point to IDs.
        cudaCatchError(hipMalloc((void **)&childl,  sizeof(int)   * (nnodes+1) * 4));

        // the following properties, for each node in the quadtree (both internal and leaf)
        cudaCatchError(hipMalloc((void **)&massl,   sizeof(float) * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&posxl,   sizeof(float) * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&posyl,   sizeof(float) * (nnodes+1)));
        // count contains the number of nested nodes for each node in quadtree
        cudaCatchError(hipMalloc((void **)&countl,  sizeof(int)   * (nnodes+1)));
        // start contains ...
        cudaCatchError(hipMalloc((void **)&startl,  sizeof(int)   * (nnodes+1)));
        cudaCatchError(hipMalloc((void **)&sortl,   sizeof(int)   * (nnodes+1)));


        cudaCatchError(hipMalloc((void **)&sourcesl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&targetsl,sizeof(int)   * (nedges)));
        cudaCatchError(hipMalloc((void **)&fxl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fyl,     sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fx_prevl,sizeof(float) * (nbodies)));
        cudaCatchError(hipMalloc((void **)&fy_prevl,sizeof(float) * (nbodies)));

        // Used for reduction in BoundingBoxKernel
        cudaCatchError(hipMalloc((void **)&maxxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&maxyl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minxl,   sizeof(float) * mp_count * FACTOR1));
        cudaCatchError(hipMalloc((void **)&minyl,   sizeof(float) * mp_count * FACTOR1));

        // Used for reduction in SpeedKernel
        cudaCatchError(hipMalloc((void **)&swgl,    sizeof(float) * mp_count * FACTOR6));
        cudaCatchError(hipMalloc((void **)&etral,   sizeof(float) * mp_count * FACTOR6));

        // Copy host data to device.
        cudaCatchError(hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));

        // cpy fx, fy , fx_prevl, fy_prevl so they are all initialized to 0 in device memory.
        cudaCatchError(hipMemcpy(fxl, fx,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fyl, fy,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fx_prevl, fx_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fy_prevl, fy_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    }

    void CUDAFA2Layout::freeGPUMemory()
    {
        hipFree(childl);

        hipFree(massl);
        hipFree(posxl);
        hipFree(posyl);
        hipFree(sourcesl);
        hipFree(targetsl);
        hipFree(countl);
        hipFree(startl);
        hipFree(sortl);

        hipFree(fxl);
        hipFree(fx_prevl);
        hipFree(fyl);
        hipFree(fy_prevl);

        hipFree(maxxl);
        hipFree(maxyl);
        hipFree(minxl);
        hipFree(minyl);

        hipFree(swgl);
        hipFree(etral);
    }

    CUDAFA2Layout::~CUDAFA2Layout()
    {
        free(mass);
        free(posx);
        free(posy);
        free(sources);
        free(targets);
        free(fx);
        free(fy);
        free(fx_prev);
        free(fy_prev);

        freeGPUMemory();
    }

    void CUDAFA2Layout::benchmark()
    {
        printf("Using %d MPs\n", mp_count);
        const int num_reps = 5;

        float times[11] = {0.0, };
        float time; // to temporarily hold time of a kernel

        const char *kernel_names[11] = {"Gravity", "Attractive", "BoundingBox", "Clear1", "TreeBuilding", "Clear2", "Summarization", "Sort", "Force", "Speed", "Displacement"};

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipDeviceSynchronize();
        auto starttime = std::chrono::high_resolution_clock::now();

        for (int i = 0; i < num_reps; ++i)
        {
            hipEventRecord(start, 0);
            GravityKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g, strong_gravity, massl, posxl, posyl, fxl, fyl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[0] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            AttractiveForceKernel<<<mp_count * FACTOR6, THREADS6>>>(nedges, posxl, posyl, massl, fxl, fyl, sourcesl, targetsl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[1] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            BoundingBoxKernel<<<mp_count * FACTOR1, THREADS1>>>(nnodes, nbodies, startl, childl, massl, posxl, posyl, maxxl, maxyl, minxl, minyl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[2] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            ClearKernel1<<<1024, 1>>>(nnodes, nbodies, childl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[3] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            TreeBuildingKernel<<<mp_count * FACTOR2, THREADS2>>>(nnodes, nbodies, childl, posxl, posyl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[4] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            ClearKernel2<<<1024, 1>>>(nnodes, startl, massl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[5] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            SummarizationKernel<<<mp_count * FACTOR3, THREADS3>>>(nnodes, nbodies, countl, childl, massl, posxl, posyl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[6] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            SortKernel<<<mp_count * FACTOR4, THREADS4>>>(nnodes, nbodies, sortl, countl, startl, childl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[7] += time;
            cudaCatchError(hipGetLastError());

            float epssq  = 0.05 * 0.05;            // Some sort of softening (eps, squared)
            float itolsq = 1.0f / (theta * theta); // Inverse tolerance, squared
            hipEventRecord(start, 0);
            ForceCalculationKernel<<<mp_count * FACTOR5, THREADS5>>>(nnodes, nbodies, itolsq, epssq, sortl, childl, massl, posxl, posyl, fxl, fyl, k_r);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[8] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            SpeedKernel<<<mp_count * FACTOR1, THREADS1>>>(nbodies, fxl, fyl, fx_prevl, fy_prevl, massl, swgl, etral);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[9] += time;
            cudaCatchError(hipGetLastError());

            hipEventRecord(start, 0);
            DisplacementKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, posxl, posyl, fxl, fyl, fx_prevl, fy_prevl);
            hipEventRecord(stop); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop);
            times[10] += time;
            cudaCatchError(hipGetLastError());
        }

        hipDeviceSynchronize(); // Not really neccesary given the preceding hipEventSynchronize(stop).
        auto endtime = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> runningtime = (endtime - starttime);
        std::chrono::microseconds runningtime_us = std::chrono::duration_cast<std::chrono::microseconds>(runningtime);

        printf("Benchmarking Results (averaging %d times):\n", num_reps);
        printf("\tkernel durations (us), resolution: ... us.:\n");
        for (int i = 0; i < 11; ++i)
        {
            printf("%s %.4f\n", kernel_names[i], 1000.0 * times[i] / (float)num_reps);
        }
        printf("\n");
        printf("Total %.2f\n", runningtime_us.count() / (float)num_reps);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void CUDAFA2Layout::doStep()
    {
        int err = 0;

        GravityKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, k_g, strong_gravity, massl, posxl, posyl, fxl, fyl);

        AttractiveForceKernel<<<mp_count * FACTOR6, THREADS6>>>(nedges, posxl, posyl, massl, fxl, fyl, sourcesl, targetsl);

        BoundingBoxKernel<<<mp_count * FACTOR1, THREADS1>>>(nnodes, nbodies, startl, childl, massl, posxl, posyl, maxxl, maxyl, minxl, minyl);

        // Build Barnes-Hut Tree
        // 1.) Set all child pointers of internal nodes (in childl) to null (-1)
        ClearKernel1<<<mp_count, 1024>>>(nnodes, nbodies, childl);
        // 2.) Build the tree
        TreeBuildingKernel<<<mp_count * FACTOR2, THREADS2>>>(nnodes, nbodies, childl, posxl, posyl);

        hipDeviceSynchronize();
        cudaCatchError(hipMemcpyFromSymbol(&err, HIP_SYMBOL(errd), sizeof(int), 0, hipMemcpyDeviceToHost));
        if (err != 0)
        {
            fprintf(stderr, "error: An error occurred in TreeBuildingKernel, errd == %d\n", err);
            // exit(EXIT_FAILURE);
        }

        // 3.) Set all cell mass values to -1.0, set all startd to null (-1)
        ClearKernel2<<<mp_count, 1024>>>(nnodes, startl, massl);

        // Recursively compute mass for each BH. cell.
        SummarizationKernel<<<mp_count * FACTOR3, THREADS3>>>(nnodes, nbodies, countl, childl, massl, posxl, posyl);

        SortKernel<<<mp_count * FACTOR4, THREADS4>>>(nnodes, nbodies, sortl, countl, startl, childl);

        // Compute repulsive forces between nodes using BH. tree.
        float epssq  = 0.05 * 0.05;            // Some sort of softening (eps, squared)
        float itolsq = 1.0f / (theta * theta); // Inverse tolerance, squared
        ForceCalculationKernel<<<mp_count * FACTOR5, THREADS5>>>(nnodes, nbodies, itolsq, epssq, sortl, childl, massl, posxl, posyl, fxl, fyl, k_r);

        hipDeviceSynchronize();
        cudaCatchError(hipMemcpyFromSymbol(&err, HIP_SYMBOL(errd), sizeof(int), 0, hipMemcpyDeviceToHost));
        if (err != 0)
        {
            fprintf(stderr, "error: An error occurred in ForceCalculationKernel, errd == %d\n", err);
            exit(EXIT_FAILURE);
        }


        SpeedKernel<<<mp_count * FACTOR1, THREADS1>>>(nbodies, fxl, fyl, fx_prevl, fy_prevl, massl, swgl, etral);

        DisplacementKernel<<<mp_count * FACTOR6, THREADS6>>>(nbodies, posxl, posyl, fxl, fyl, fx_prevl, fy_prevl);

        iteration++;
    }

    void CUDAFA2Layout::doSteps(int n)
    {
        for (int i = 0; i < n; ++i) doStep();
    }

    void CUDAFA2Layout::setScale(float s)
    {
        k_r = s;
    }

    void CUDAFA2Layout::setGravity(float g)
    {
        k_g = g;
    }

    void CUDAFA2Layout::retrieveLayoutFromGPU()
    {
        hipDeviceSynchronize();
        cudaCatchError(hipMemcpy(posx, posxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
        cudaCatchError(hipMemcpy(posy, posyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
        cudaCatchError(hipMemcpy(fx, fxl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
        cudaCatchError(hipMemcpy(fy, fyl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
        cudaCatchError(hipMemcpy(fx_prev, fx_prevl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
        cudaCatchError(hipMemcpy(fy_prev, fy_prevl, sizeof(float) * nbodies, hipMemcpyDeviceToHost));
    }

    void CUDAFA2Layout::sendLayoutToGPU()
    {
        hipDeviceSynchronize();
        cudaCatchError(hipMemcpy(posxl, posx, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(posyl, posy, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fxl, fx,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fyl, fy,           sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fx_prevl, fx_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(fy_prevl, fy_prev, sizeof(float) * nbodies, hipMemcpyHostToDevice));
    }

    void CUDAFA2Layout::sendGraphToGPU()
    {
        hipDeviceSynchronize();
        cudaCatchError(hipMemcpy(massl, mass, sizeof(float) * nbodies, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(sourcesl, sources, sizeof(int) * nedges, hipMemcpyHostToDevice));
        cudaCatchError(hipMemcpy(targetsl, targets, sizeof(int) * nedges, hipMemcpyHostToDevice));
    }


    void CUDAFA2Layout::writeToPNG(const int width, const int height, const char *path)
    {
        hipDeviceSynchronize(); // Wait for all kernels to complete.
        ComputeLayoutDimensions<<<mp_count * FACTOR1, THREADS1>>>(nbodies, posxl, posyl, maxxl, maxyl, minxl, minyl);

        // Retrieve data form GPU
        float minx_h, maxx_h, miny_h, maxy_h;
        hipDeviceSynchronize();
        cudaCatchError(hipMemcpyFromSymbol(&minx_h, HIP_SYMBOL(minxdg), sizeof(float)));
        cudaCatchError(hipMemcpyFromSymbol(&maxx_h, HIP_SYMBOL(maxxdg), sizeof(float)));
        cudaCatchError(hipMemcpyFromSymbol(&miny_h, HIP_SYMBOL(minydg), sizeof(float)));
        cudaCatchError(hipMemcpyFromSymbol(&maxy_h, HIP_SYMBOL(maxydg), sizeof(float)));
        cudaCatchError(hipMemcpy(posx, posxl,     sizeof(float) * nbodies, hipMemcpyDeviceToHost));
        cudaCatchError(hipMemcpy(posy, posyl,     sizeof(float) * nbodies, hipMemcpyDeviceToHost));
        float img_width = 5000;
        float img_height = 5000;

        const float xRange = maxx_h - minx_h;
        const float yRange = maxy_h - miny_h;
        const float xCenter = minx_h + xRange / 2.0;
        const float yCenter = miny_h + yRange / 2.0;
        const float minX = xCenter - xRange   / 2.0;
        const float minY = yCenter - yRange   / 2.0;
        const float xScale = img_width/xRange;
        const float yScale = img_height/yRange;

        // Here we need to do some guessing as to what the optimal
        // opacity of nodes and edges might be, given how many of them we need to draw.
        const float node_opacity = 1/(0.0001  * graph.num_nodes());
        const float edge_opacity = 1/(0.00001 * graph.num_edges());


        pngwriter layout_png(img_width, img_height, 0, path);
        layout_png.invert(); // set bg. to white.

        for (int n1 = 0; n1 < graph.num_nodes(); ++n1)
        {
            // Plot node,
            layout_png.filledcircle_blend((posx[n1] - minX)*xScale,
                                          (posy[n1] - minY)*yScale,
                                          3, node_opacity, 0, 0, 0);
            for (nid_t n2 : graph.neighbors_with_geq_id(n1)) {
                // ... and edge.
                layout_png.line_blend((posx[n1] - minX)*xScale, (posy[n1] - minY)*yScale,
                                      (posx[n2] - minX)*xScale, (posy[n2] - minY)*yScale,
                                      edge_opacity, 0, 0, 0);
            }
        }
        // Write to file.
        layout_png.write_png();
    }
}
